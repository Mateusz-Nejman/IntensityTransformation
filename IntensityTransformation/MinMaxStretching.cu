#include "hip/hip_runtime.h"
#include "MinMaxStretching.cuh"
#include <array>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

__global__ void stretchKernel(unsigned char* output, unsigned char* input, unsigned char* values, int count)
{
    unsigned int a = blockDim.x * blockIdx.x + threadIdx.x;
    output[a] = values[(int)input[a]];
}

__global__ void minMaxKernel(unsigned char* input, int count, unsigned char* _minMax)
{
    unsigned int a = blockIdx.x * blockDim.x + threadIdx.x;

    if (_minMax[0] > input[a])
    {
        _minMax[0] = input[a];
    }
    else if (_minMax[1] < input[a])
    {
        _minMax[1] = input[a];
    }
}

__global__ void calculateDifferenceKernel(unsigned char* _minMax, float* difference)
{
    *difference = 255.0f / (_minMax[1] - _minMax[0]);
}

__global__ void stretchValuesKernel(unsigned char* values, unsigned char* _min, float* difference)
{
    unsigned int a = blockIdx.x * blockDim.x + threadIdx.x;
    values[a] = (a - _min[0]) * (*difference);
}

MinMaxStretching::MinMaxStretching(unsigned char* values, int count, bool fromGpuValues)
{
    _count = count;
    _threadsPerBlock = 1024;
    _blockCount = _count / _threadsPerBlock;

    hipMalloc(&_values, _count * sizeof(unsigned char));
    hipMemcpy(_values, values, count * sizeof(unsigned char), fromGpuValues ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice);
}

MinMaxStretching::~MinMaxStretching()
{
  hipFree(gpuDarkestBrightest);
  hipFree(gpuDifference);
  hipFree(gpuOutput);
  hipFree(gpuStretchValues);
  hipFree(_values);
}

unsigned char* MinMaxStretching::stretch()
{
    auto gpuOutput = stretchGpu();
    return gpuOutput.getData();
}

MinMaxStretching MinMaxStretching::stretchGpu()
{
    float* gpuDifference;
    unsigned char* gpuDarkestBrightest;
    unsigned char* gpuStretchValues;
    unsigned char* gpuOutput;

    //Allocating & filling
    hipMalloc(&gpuDarkestBrightest, 2 * sizeof(unsigned char));
    hipMemcpy(gpuDarkestBrightest, _values, 2 * sizeof(unsigned char), hipMemcpyDeviceToDevice);
    hipMalloc(&gpuDifference, sizeof(float));
    hipMalloc(&gpuOutput, _count * sizeof(unsigned char));

    //GPU calculations
    minMaxKernel << <_blockCount, _threadsPerBlock >> > (_values, _count, gpuDarkestBrightest);
    calculateDifferenceKernel << <1, 1 >> > (gpuDarkestBrightest, gpuDifference);
    hipMalloc(&gpuStretchValues, 256 * sizeof(unsigned char));
    stretchValuesKernel << <1, _threadsPerBlock >> > (gpuStretchValues, gpuDarkestBrightest, gpuDifference);
    stretchKernel << <_blockCount, _threadsPerBlock >> > (gpuOutput, _values, gpuStretchValues, _count);

    //Free GPU memory
    //hipFree(gpuDifference);
    //hipFree(gpuDarkestBrightest);
    //hipFree(gpuStretchValues);

    return MinMaxStretching(gpuOutput, _count, true);
}

void MinMaxStretching::minMaxToCheck()
{
  //Allocating & filling
  hipMalloc(&gpuDarkestBrightest, 2 * sizeof(unsigned char));
  hipMemcpy(gpuDarkestBrightest, _values, 2 * sizeof(unsigned char), hipMemcpyDeviceToDevice);
  hipMalloc(&gpuDifference, sizeof(float));
  hipMalloc(&gpuOutput, _count * sizeof(unsigned char));
  minMaxKernel << <_blockCount, _threadsPerBlock >> > (_values, _count, gpuDarkestBrightest);
  calculateDifferenceKernel << <1, 1 >> > (gpuDarkestBrightest, gpuDifference);
}

unsigned char* MinMaxStretching::getData()
{
  unsigned char* data = new unsigned char[_count];
  hipMemcpy(data,_values,_count * sizeof(unsigned char), hipMemcpyDeviceToHost);
  return data;
}